// SPDX-License-Identifier: GPLv3-or-later
// Copyright © 2020 Or Toledano
#include "cudamultiply.cuh"
#include <hipblas.h>
#include <cstdio>


#define M 4
#define MAT_MULT_ITERS 64

#define IDX2C(i, j, ld) (((j)*(ld))+(i))
#define IDX(matrix, row, column) ((M*M)*(matrix)+(IDX2C((row),(column),M)))



// Multiply the ind0 matrix with the ind1 matrix, add to the ind_res matrix
// assume all indices are different and matrices are initialized
void
cuda_multiply_add(const unsigned char *cube_src, unsigned char *cube_dst, int ind0,
             int ind1, int ind_res) {
    float *devPtrA, *devPtrB, *devPtrRes;
    // Error codes for alloc on gpu
    hipError_t cudaStat1 = hipMalloc((float **) &devPtrA,
                                       M * M * sizeof(float));
    hipError_t cudaStat2 = hipMalloc((float **) &devPtrB,
                                       M * M * sizeof(float));
    hipError_t cudaStat3 = hipMalloc((float **) &devPtrRes,
                                       M * M * sizeof(float));
    if (cudaStat1 != hipSuccess || cudaStat2 != hipSuccess ||
        cudaStat3 != hipSuccess) {
        printf("device memory allocation failed\n");
        exit(EXIT_FAILURE);
    }
    hipblasHandle_t handle;
    hipblasStatus_t stat1 = hipblasCreate(&handle);
    // Error codes for CUBLAS initialization
    if (stat1 != HIPBLAS_STATUS_SUCCESS) {
        printf("CuBLAS initialization failed\n");
        exit(EXIT_FAILURE);
    }
    // Download from cpu to gpu
    stat1 = hipblasSetMatrix(M, M, sizeof(*cube_src), cube_src + M * M * ind0, M,
                            devPtrA, M);
    hipblasStatus_t stat2 = hipblasSetMatrix(M, M, sizeof(*cube_src),
                                           cube_src + M * M * ind1, M, devPtrA,
                                           M);
    // Error codes for cpu to gpu bus
    if (stat1 != HIPBLAS_STATUS_SUCCESS || stat2 != HIPBLAS_STATUS_SUCCESS) {
        printf("data download failed\n");
        hipFree(devPtrA);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }

    const float alpha = 1;
    const float beta = 1;

    // Single precision real matrices multiplication
    hipblasStatus_t statMult = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M,
                                          M, M, &alpha, devPtrA, M, devPtrB, M,
                                          &beta, devPtrRes, M);
    if (statMult != HIPBLAS_STATUS_SUCCESS) {
        printf("CuBLAS gemm failed\n");
        exit(EXIT_FAILURE);
    }

    hipblasStatus_t statRes = hipblasGetMatrix(M, M, sizeof(float), devPtrA, M,
                                             cube_dst + M * M * ind_res, M);
    // Error codes for gpu to cpu bus
    if (statRes != HIPBLAS_STATUS_SUCCESS) {
        printf("data upload failed\n");
        hipFree(devPtrA);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }

    hipblasDestroy(handle);
    hipFree(devPtrRes);
    hipFree(devPtrB);
    hipFree(devPtrA);
    exit(EXIT_SUCCESS);
}